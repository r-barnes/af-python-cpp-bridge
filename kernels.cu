#include "hip/hip_runtime.h"
#include <stdexcept>
#include <string>
#include <arrayfire.h>
#include <iostream>
#include <af/hip/hip_runtime.h>
#include <hip/hip_fp16.h>
typedef float2 hipFloatComplex;
typedef hipFloatComplex cfloat;

__global__
void absify_cuda(float *amp, cfloat const *const spectrum, const int n)
{
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < n; i += blockDim.x * gridDim.x){
        amp[i] = hypotf(spectrum[i].x, spectrum[i].y);
    }
}

void absify(float *amp, af::cfloat const *const spectrum, const int n){
  int const af_id             = af::getDevice();
  int const cuda_id           = afcu::getNativeId(af_id);
  hipStream_t af_cuda_stream = afcu::getStream(cuda_id);

  std::cout<<"Absify on "<<amp<<" and "<<spectrum<<std::endl;
  absify_cuda<<<16384,256,0,af_cuda_stream>>>(amp, reinterpret_cast<cfloat const *>(spectrum), n);
  const auto ret = hipDeviceSynchronize();
  if(ret!=hipSuccess){
    throw std::runtime_error(std::string("Kernel absify failed! ") + hipGetErrorString(hipGetLastError()));
  }
}



#define __cabsf(in) hypotf(in.x, in.y)

__device__ cfloat __cplx2f(float x, float y) {
    cfloat res = {x, y};
    return res;
}

__device__ cfloat __cdivf(cfloat lhs, cfloat rhs) {
    // Normalize by absolute value and multiply
    float rhs_abs     = __cabsf(rhs);
    float inv_rhs_abs = 1.0f / rhs_abs;
    float rhs_x       = inv_rhs_abs * rhs.x;
    float rhs_y       = inv_rhs_abs * rhs.y;
    cfloat out = {lhs.x * rhs_x + lhs.y * rhs_y, lhs.y * rhs_x - lhs.x * rhs_y};
    out.x *= inv_rhs_abs;
    out.y *= inv_rhs_abs;
    return out;
}

__device__ cfloat __cmulf(cfloat lhs, cfloat rhs) {
    cfloat out;
    out.x = lhs.x * rhs.x - lhs.y * rhs.y;
    out.y = lhs.x * rhs.y + lhs.y * rhs.x;
    return out;
}


__global__
void rescale_cuda(float amp_max, cfloat *spectrum, cfloat const *const amp_meas, float const *const amp, const int n)
{
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < n; i += blockDim.x * gridDim.x){
    float const maxv = fmax(amp_max, amp[i]);
    cfloat val5 = __cdivf(spectrum[i], __cplx2f(maxv, 0));
    cfloat val7 = __cmulf(val5, amp_meas[i]);
    spectrum[i] = val7;
  }
}

void rescale(float amp_max, af::cfloat *spectrum, af::cfloat const *const amp_meas, float const *const amp, const int n){
  int const af_id             = af::getDevice();
  int const cuda_id           = afcu::getNativeId(af_id);
  hipStream_t af_cuda_stream = afcu::getStream(cuda_id);

  rescale_cuda<<<16384,256,0,af_cuda_stream>>>(amp_max, 
    reinterpret_cast<cfloat *>(spectrum),
    reinterpret_cast<cfloat const *>(amp_meas),
    amp,
    n
  );
  const auto ret = hipDeviceSynchronize();
  if(ret!=hipSuccess){
    throw std::runtime_error(std::string("Kernel rescale failed! ") + hipGetErrorString(hipGetLastError()));
  }
}
